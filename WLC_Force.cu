#include "System.h"
#include "WLC_Force.h" 
#include "functor_wlc.h"

/*
the structure of lengthZero_index is 
0  1  2  3 
4  5  6  7 
8  9  10 11
12 13 14 15 for a 4 node system. 
index/4 = row,
index%4 = col. If you apply force to column node always or row node always then 
each thread will apply opposing forces to springs. 
if you decide to apply force to column instead of rows, you'll need sign change
LengthZero_value is symmetric, so values line up correctly.
*/

void WLC_Force(
	NodeInfoVecs& nodeInfoVecs,
	WLCInfoVecs& wlcInfoVecs,  
	GeneralParams& generalParams) {
 
 
	thrust::counting_iterator<unsigned> startEdgeIter(0);
			  
	//
	thrust::for_each( 
		thrust::make_zip_iterator( 
			thrust::make_tuple(startEdgeIter,
								nodeInfoVecs.isNodeFixed.begin() )),
		thrust::make_zip_iterator(
			thrust::make_tuple(startEdgeIter,
								nodeInfoVecs.isNodeFixed.begin() )) + generalParams.maxNodeCount,
		functor_wlc(
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeForceX.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeForceY.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeForceZ.data()),
 
			generalParams.kB,
			generalParams.persistenceLengthMon,
			generalParams.CLM,
			generalParams.temperature,
			generalParams.maxNeighborCount,
			generalParams.maxNodeCount,
			generalParams.nummonfiberarea,

			thrust::raw_pointer_cast(wlcInfoVecs.lengthZero.data()),
			thrust::raw_pointer_cast(wlcInfoVecs.globalNeighbors.data()),
			thrust::raw_pointer_cast(wlcInfoVecs.currentNodeEdgeCountVector.data()),
			thrust::raw_pointer_cast(wlcInfoVecs.numOriginalNeighborsNodeVector.data()) ) );
};

