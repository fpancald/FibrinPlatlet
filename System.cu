#include "hip/hip_runtime.h"
#include "Storage.h"
#include "Link_Nodes.h"
#include "WLC_Force.h"
#include "Torsion_Force.h"
#include "Plt_Arm_Node_Force.h"
#include "Plt_Arm_Plt_Force.h"
#include "Plt_Field_Node_Force.h"
#include "Plt_Field_Plt_Force.h"
#include "Plt_Vol_Exc_Force.h"

#include "Params_Calc.h"
#include "Advance_Positions.h"
#include "Bucket_Net.h"
#include "Bucket_Plt.h"
#include "System.h"



void System::setBucketScheme() {

	init_dim_general(
		nodeInfoVecs,
		pltInfoVecs,
		domainParams,
		auxVecs,
		generalParams);

	init_net_inct_bucket(
		nodeInfoVecs,
		pltInfoVecs,
		domainParams,
		auxVecs,
		generalParams);

	build_net_inct_bucket(
		nodeInfoVecs,
		pltInfoVecs,
		domainParams,
		auxVecs,
		generalParams);

	extend_net_inct_bucket(
		nodeInfoVecs,
		pltInfoVecs,
		domainParams,
		auxVecs,
		generalParams);
		
	init_plt_inct_bucket(
		nodeInfoVecs,
		pltInfoVecs,
		domainParams,
		auxVecs,
		generalParams);

	build_plt_inct_bucket(
		nodeInfoVecs,
		pltInfoVecs,
		domainParams,
		auxVecs,
		generalParams);

	extend_plt_inct_bucket(
		nodeInfoVecs,
		pltInfoVecs,
		domainParams,
		auxVecs,
		generalParams);
		
};

void System::solveForces() {

	//RESET FORCE TO ZERO AT BEGINNING/////////////////////////////////////////////////
	thrust::fill(nodeInfoVecs.nodeForceX.begin(),nodeInfoVecs.nodeForceX.end(),0);
	thrust::fill(nodeInfoVecs.nodeForceY.begin(),nodeInfoVecs.nodeForceY.end(),0);
	thrust::fill(nodeInfoVecs.nodeForceZ.begin(),nodeInfoVecs.nodeForceZ.end(),0);
	
	thrust::fill(pltInfoVecs.pltForceX.begin(),pltInfoVecs.pltForceX.end(),0);
	thrust::fill(pltInfoVecs.pltForceY.begin(),pltInfoVecs.pltForceY.end(),0);
	thrust::fill(pltInfoVecs.pltForceZ.begin(),pltInfoVecs.pltForceZ.end(),0);

	
	if (generalParams.linking == true) {
		Link_Nodes(
			nodeInfoVecs,
			wlcInfoVecs,
			auxVecs,
			generalParams);
	}
	Torsion_Force(nodeInfoVecs, torsionInfoVecs, generalParams);

	//std::cout<<"prewlc"<<std::endl;
	WLC_Force(nodeInfoVecs, wlcInfoVecs, generalParams);

	//platetelet-node forces
	//RESETS PLATELET FORCES
	if (generalParams.pltfrcfld == true) {// note: this force-field includes both pulling and pushing
		Plt_Field_Node_Force(//platelet on node force field
			nodeInfoVecs,
			wlcInfoVecs,
			generalParams,
			pltInfoVecs,
			auxVecs);
		if (generalParams.pltonplt == true) {
			Plt_Field_Plt_Force(//platelet on platelet interaction through force field
				generalParams,
				pltInfoVecs,
				auxVecs);
		}

	}
	else if (generalParams.plttndrl == true) { //note for now force-field type has priority over tndrl-type

		// Tndrl-node pulling
		Plt_Arm_Node_Force(
		  nodeInfoVecs,
		  wlcInfoVecs,
		  generalParams,
		  pltInfoVecs,
		  auxVecs);

		//Tndrl-Plt pulling
		if (generalParams.pltonplt == true) {
			/*Plt_Arm_Plt_Force(//platelet on platelet interaction through tndrl
				generalParams,
				pltInfoVecs,
				auxVecs);*/
		}

		Plt_Vol_Exc_Force(//push for volume exclusion
			nodeInfoVecs,
			wlcInfoVecs,
			generalParams,
			pltInfoVecs,
			auxVecs);

	}




};


void System::solveSystem() {

	//set initial bucket scheme
	setBucketScheme();

	//set initial epsilon
	generalParams.epsilon = (1.0) *
		sqrt(6.0*generalParams.kB * generalParams.temperature * generalParams.dtTemp / generalParams.viscousDamp_Fibrin);

	while (generalParams.runSim == true) {

		generalParams.iterationCounter++;
		generalParams.currentTime += generalParams.dtTemp;
		std::cout<<"iterationCount: "<< generalParams.iterationCounter <<std::endl;

		Advance_Positions(
			nodeInfoVecs,
			pltInfoVecs,
		 	generalParams);	
		
		if (generalParams.iterationCounter % 10 == 0) {
			setBucketScheme();
		}

		
		solveForces(); //resets and solves forces for next time step


		if (generalParams.iterationCounter % 500 == 0) {

			storage->print_VTK_File();
			//store sum of all forces on each node. Used in stress calculations
			//store before upadting storage class.

			//WARNING BEFORE CALLING SAVE_PARAMS CALCULATE THEM FIRST
			Params_Calc(
    			wlcInfoVecs,
    			nodeInfoVecs,
    			generalParams,
    			pltInfoVecs);

			storage->save_params();

			generalParams.epsilon = (1.0) *
				sqrt(6.0 * generalParams.kB * generalParams.temperature * generalParams.dtTemp / generalParams.viscousDamp_Fibrin);

		}

	}

};

System::System()  {};

void System::assignStorage(std::shared_ptr<Storage> _storage) {
	storage = _storage;
}

//__host__ __device__
void System::initializeSystem(
	thrust::host_vector<bool>& hostIsNodeFixed,
	thrust::host_vector<double>& hostPosX,
	thrust::host_vector<double>& hostPosY,
	thrust::host_vector<double>& hostPosZ,
	thrust::host_vector<unsigned>& hostWLCEdgeLeft,
	thrust::host_vector<unsigned>& hostWLCEdgeRight,
	thrust::host_vector<double>& hostWLCLenZero,

	thrust::host_vector<unsigned>& hostWLCSubEdgeLeft,
	thrust::host_vector<unsigned>& hostWLCSubEdgeRight,
	thrust::host_vector<double>& hostWLCSubLenZero,
	thrust::host_vector<unsigned>& hostTorsionIndexLeft,
	thrust::host_vector<unsigned>& hostTorsionIndexCenter,
	thrust::host_vector<unsigned>& hostTorsionIndexRight,
	thrust::host_vector<double>& hostTorsionAngleZero,
	//platelets
	thrust::host_vector<bool>& hostIsPltFixed,
	thrust::host_vector<double>& hostPltPosX,
	thrust::host_vector<double>& hostPltPosY,
	thrust::host_vector<double>& hostPltPosZ) {

	std::cout<< "total Edge Count: "<< generalParams.originEdgeCount << std::endl;
	std::cout << "max num nodes: " << generalParams.maxNodeCount << std::endl;
	//platelets

	std::cout << "max num platelets in device: " << generalParams.maxPltCount << std::endl;



	setPltVecs(
		hostIsPltFixed,
		hostPltPosX,
		hostPltPosY,
		hostPltPosZ);

	setNodeVecs(//calls initDimensionBucketScheme
		hostIsNodeFixed,
		hostPosX,
		hostPosY,
		hostPosZ);

	setTorsionVecs(
		hostTorsionIndexLeft,
		hostTorsionIndexCenter,
		hostTorsionIndexRight,
		hostTorsionAngleZero);

	setWLCVecs(
		hostWLCEdgeLeft,
		hostWLCEdgeRight,
		hostWLCLenZero );

		
};

void System::setNodeVecs(
	thrust::host_vector<bool>& hostIsNodeFixed,
	thrust::host_vector<double>& hostPosX,
	thrust::host_vector<double>& hostPosY,
	thrust::host_vector<double>& hostPosZ) {


	nodeInfoVecs.sumForcesOnNode.resize(generalParams.maxNodeCount);

	nodeInfoVecs.nodeVelocity.resize(generalParams.maxNodeCount);

	nodeInfoVecs.nodeLocX.resize(generalParams.maxNodeCount);
	nodeInfoVecs.nodeLocY.resize(generalParams.maxNodeCount);
	nodeInfoVecs.nodeLocZ.resize(generalParams.maxNodeCount);

	nodeInfoVecs.nodeForceX.resize(generalParams.maxNodeCount);
	nodeInfoVecs.nodeForceY.resize(generalParams.maxNodeCount);
	nodeInfoVecs.nodeForceZ.resize(generalParams.maxNodeCount);

	nodeInfoVecs.discretizedEdgeStrain.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);
	nodeInfoVecs.discretizedEdgeAlignment.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);

	//sized larger for input later
	nodeInfoVecs.deviceEdgeLeft.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);
	nodeInfoVecs.deviceEdgeRight.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);


	thrust::fill(nodeInfoVecs.discretizedEdgeStrain.begin(), nodeInfoVecs.discretizedEdgeStrain.end(),0.0);
	thrust::fill(nodeInfoVecs.deviceEdgeRight.begin(), nodeInfoVecs.deviceEdgeRight.end(), 0);	//fill force and velocity with zeros for computation.
	thrust::fill(nodeInfoVecs.deviceEdgeLeft.begin(), nodeInfoVecs.deviceEdgeLeft.end(), 0);	//fill force and velocity with zeros for computation.

	thrust::fill(nodeInfoVecs.sumForcesOnNode.begin(), nodeInfoVecs.sumForcesOnNode.end(), 0);


	thrust::copy(hostPosX.begin(), hostPosX.end(), nodeInfoVecs.nodeLocX.begin());
	thrust::copy(hostPosY.begin(), hostPosY.end(), nodeInfoVecs.nodeLocY.begin());
	thrust::copy(hostPosZ.begin(), hostPosZ.end(), nodeInfoVecs.nodeLocZ.begin());


	//copy fixed positions
	nodeInfoVecs.isNodeFixed.resize(generalParams.maxNodeCount);
	thrust::copy(hostIsNodeFixed.begin(), hostIsNodeFixed.end(), nodeInfoVecs.isNodeFixed.begin());

	nodeInfoVecs.isNodeInPltVol.resize(generalParams.maxNodeCount);
	thrust::fill(nodeInfoVecs.isNodeInPltVol.begin(),nodeInfoVecs.isNodeInPltVol.end(),false);

	nodeInfoVecs.linksThreadMade.resize(generalParams.maxNodeCount);
	nodeInfoVecs.delinksThreadMade.resize(generalParams.maxNodeCount);
	nodeInfoVecs.idMadeTempLeft.resize(generalParams.maxNodeCount * generalParams.maxLinksPerIteration);
	nodeInfoVecs.idMadeTempRight.resize(generalParams.maxNodeCount * generalParams.maxLinksPerIteration);

	//at this point all nodes are filled, so we can generate domainParams
	init_dim_general(
		nodeInfoVecs,
		pltInfoVecs,
		domainParams,
		auxVecs,
		generalParams);


	domainParams.originMinX = domainParams.minX;
	domainParams.originMaxX = domainParams.maxX;
	domainParams.originMinY = domainParams.minY;
	domainParams.originMaxY = domainParams.maxY;
	domainParams.originMinZ = domainParams.minZ;
	domainParams.originMaxZ = domainParams.maxZ;

	std::cout<< "node count : " <<nodeInfoVecs.nodeLocY.size()<< std::endl;


	auxVecs.id_bucket_net_intc.resize(generalParams.maxNodeCount);
	auxVecs.id_value_net_intc.resize(generalParams.maxNodeCount);
	auxVecs.id_bucket_expanded_net_intc.resize(27 * (generalParams.maxNodeCount));
	auxVecs.id_value_expanded_net_intc.resize(27 *( generalParams.maxNodeCount ));

	
	auxVecs.id_bucket_plt_intc.resize(generalParams.maxNodeCount);
	auxVecs.id_value_plt_intc.resize(generalParams.maxNodeCount);
	auxVecs.id_bucket_expanded_plt_intc.resize(27 * (generalParams.maxNodeCount));
	auxVecs.id_value_expanded_plt_intc.resize(27 *( generalParams.maxNodeCount ));

	
	auxVecs.idPlt_bucket.resize(generalParams.maxPltCount);
	auxVecs.idPlt_value.resize(generalParams.maxPltCount);
	auxVecs.idPlt_bucket_expanded.resize(27 * (generalParams.maxPltCount));
	auxVecs.idPlt_value_expanded.resize(27 *( generalParams.maxPltCount ));

};

//platelet
void System::setPltVecs(
	thrust::host_vector<bool>& hostIsPltFixed,
	thrust::host_vector<double>& hostPltPosX,
	thrust::host_vector<double>& hostPltPosY,
	thrust::host_vector<double>& hostPltPosZ) {


	pltInfoVecs.sumForcesOnPlt.resize(generalParams.maxPltCount);

	pltInfoVecs.pltVelocity.resize(generalParams.maxPltCount);

	pltInfoVecs.pltLocX.resize(generalParams.maxPltCount);
	pltInfoVecs.pltLocY.resize(generalParams.maxPltCount);
	pltInfoVecs.pltLocZ.resize(generalParams.maxPltCount);

	pltInfoVecs.pltForceX.resize(generalParams.maxPltCount);
	pltInfoVecs.pltForceY.resize(generalParams.maxPltCount);
	pltInfoVecs.pltForceZ.resize(generalParams.maxPltCount);

	pltInfoVecs.pltImagingConnection.resize(generalParams.maxPltCount * generalParams.plt_tndrl_intrct);
	pltInfoVecs.nodeImagingConnection.resize(generalParams.maxPltCount * generalParams.plt_tndrl_intrct);

	pltInfoVecs.nodeUnreducedId.resize(generalParams.maxPltCount * generalParams.plt_other_intrct);
	pltInfoVecs.nodeUnreducedForceX.resize(generalParams.maxPltCount * generalParams.plt_other_intrct);
	pltInfoVecs.nodeUnreducedForceY.resize(generalParams.maxPltCount * generalParams.plt_other_intrct);
	pltInfoVecs.nodeUnreducedForceZ.resize(generalParams.maxPltCount * generalParams.plt_other_intrct);

	pltInfoVecs.nodeReducedId.resize(generalParams.maxPltCount * generalParams.plt_other_intrct);
	pltInfoVecs.nodeReducedForceX.resize(generalParams.maxPltCount * generalParams.plt_other_intrct);
	pltInfoVecs.nodeReducedForceY.resize(generalParams.maxPltCount * generalParams.plt_other_intrct);
	pltInfoVecs.nodeReducedForceZ.resize(generalParams.maxPltCount * generalParams.plt_other_intrct);

	thrust::fill(pltInfoVecs.sumForcesOnPlt.begin(), pltInfoVecs.sumForcesOnPlt.end(), 0);


	thrust::copy(hostPltPosX.begin(), hostPltPosX.end(), pltInfoVecs.pltLocX.begin());
	thrust::copy(hostPltPosY.begin(), hostPltPosY.end(), pltInfoVecs.pltLocY.begin());
	thrust::copy(hostPltPosZ.begin(), hostPltPosZ.end(), pltInfoVecs.pltLocZ.begin());


	std::cout<<"num platelets: "<< pltInfoVecs.pltLocX.size() << std::endl;
	std::cout<<"num platelets var: "<< generalParams.maxPltCount << std::endl;
	//copy fixed positions
	pltInfoVecs.isPltFixed.resize(generalParams.maxPltCount);
	thrust::fill(pltInfoVecs.isPltFixed.begin(), pltInfoVecs.isPltFixed.end(), false);
	//thrust::copy(hostIsPltFixed.begin(), hostIsPltFixed.end(), pltInfoVecs.isPltFixed.begin());


	auxVecs.idPlt_bucket.resize(generalParams.maxPltCount);
	auxVecs.idPlt_value.resize(generalParams.maxPltCount);
	auxVecs.idPlt_bucket_expanded.resize(27 *( generalParams.maxPltCount ));
	auxVecs.idPlt_value_expanded.resize(27 * (generalParams.maxPltCount));

    pltInfoVecs.tndrlNodeId.resize(generalParams.maxPltCount * generalParams.plt_tndrl_intrct);
	pltInfoVecs.tndrlNodeType.resize(generalParams.maxPltCount * generalParams.plt_tndrl_intrct);

	//fill with flag vales.	
	std::cout<<"maxIdFlag: "<< generalParams.maxIdCountFlag<<std::endl;

	thrust::fill(pltInfoVecs.tndrlNodeId.begin(),pltInfoVecs.tndrlNodeId.end(), generalParams.maxIdCountFlag);
	
	thrust::fill(pltInfoVecs.tndrlNodeType.begin(),pltInfoVecs.tndrlNodeType.end(), 0);
  
};

void System::setTorsionVecs(
	thrust::host_vector<unsigned>& hostTorsionIndexLeft,
	thrust::host_vector<unsigned>& hostTorsionIndexCenter,
	thrust::host_vector<unsigned>& hostTorsionIndexRight,
	thrust::host_vector<double>& hostTorsionAngleZero) {


	torsionInfoVecs.leftIndex.resize(generalParams.totalTorsionCount);
	torsionInfoVecs.centerIndex.resize(generalParams.totalTorsionCount);
	torsionInfoVecs.rightIndex.resize(generalParams.totalTorsionCount);
	torsionInfoVecs.angleZero.resize(generalParams.totalTorsionCount);

	thrust::copy(hostTorsionIndexLeft.begin(), hostTorsionIndexLeft.end(), torsionInfoVecs.leftIndex.begin());
	thrust::copy(hostTorsionIndexCenter.begin(), hostTorsionIndexCenter.end(), torsionInfoVecs.centerIndex.begin());
	thrust::copy(hostTorsionIndexRight.begin(), hostTorsionIndexRight.end(), torsionInfoVecs.rightIndex.begin());

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				torsionInfoVecs.leftIndex.begin(),
				torsionInfoVecs.centerIndex.begin(),
				torsionInfoVecs.rightIndex.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				torsionInfoVecs.leftIndex.begin(),
				torsionInfoVecs.centerIndex.begin(),
				torsionInfoVecs.rightIndex.begin())) + generalParams.totalTorsionCount,
			torsionInfoVecs.angleZero.begin(),//save vector
		TorsionAngleFunctor(
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data())));

	torsionInfoVecs.forceX.resize(torsionInfoVecs.factor * generalParams.totalTorsionCount);
	torsionInfoVecs.forceY.resize(torsionInfoVecs.factor * generalParams.totalTorsionCount);
	torsionInfoVecs.forceZ.resize(torsionInfoVecs.factor * generalParams.totalTorsionCount);
	torsionInfoVecs.tempForceX.resize(torsionInfoVecs.factor * generalParams.totalTorsionCount);
	torsionInfoVecs.tempForceY.resize(torsionInfoVecs.factor * generalParams.totalTorsionCount);
	torsionInfoVecs.tempForceZ.resize(torsionInfoVecs.factor * generalParams.totalTorsionCount);

	thrust::fill(torsionInfoVecs.forceX.begin(), torsionInfoVecs.forceX.end(), 0.0);
	thrust::fill(torsionInfoVecs.forceY.begin(), torsionInfoVecs.forceY.end(), 0.0);
	thrust::fill(torsionInfoVecs.forceZ.begin(), torsionInfoVecs.forceZ.end(), 0.0);

	torsionInfoVecs.tempTorIndices.resize(torsionInfoVecs.factor * generalParams.totalTorsionCount);
	torsionInfoVecs.reducedIds.resize(torsionInfoVecs.factor * generalParams.totalTorsionCount);
};

void System::setWLCVecs(
	thrust::host_vector<unsigned>& hostWLCSubEdgeLeft,
	thrust::host_vector<unsigned>& hostWLCSubEdgeRight,
	thrust::host_vector<double>& hostWLCSubLenZero ) {

	wlcInfoVecs.globalNeighbors.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);
	wlcInfoVecs.currentNodeEdgeCountVector.resize(generalParams.maxNodeCount);

	wlcInfoVecs.lengthZero.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);
	wlcInfoVecs.numOriginalNeighborsNodeVector.resize(generalParams.maxNodeCount);

	//default value is maxNodeCount
	thrust::fill(wlcInfoVecs.globalNeighbors.begin(), wlcInfoVecs.globalNeighbors.end(), generalParams.maxNodeCount);
	thrust::fill(wlcInfoVecs.currentNodeEdgeCountVector.begin(), wlcInfoVecs.currentNodeEdgeCountVector.end(),0);
	thrust::fill(wlcInfoVecs.lengthZero.begin(), wlcInfoVecs.lengthZero.end(), 0.0);



	nodeInfoVecs.deviceEdgeLeft = hostWLCSubEdgeLeft;
	nodeInfoVecs.deviceEdgeRight = hostWLCSubEdgeRight;

	//scan through hostAdj and put in device.
	for (unsigned id = 0; id < hostWLCSubLenZero.size(); id++) {

		unsigned idL = hostWLCSubEdgeLeft[id];
		unsigned idR = hostWLCSubEdgeRight[id];

		double edgeLen = hostWLCSubLenZero[id];
		//we use the lengthZero vector to identify edges as well.
		//node id is row, column node is connected to row node.

		//add edge for left node
		unsigned edgeNumL = wlcInfoVecs.currentNodeEdgeCountVector[idL]; //number of edges on (nodeId = row)	is that entry in cECV
		unsigned indexL = idL*generalParams.maxNeighborCount + edgeNumL;
		wlcInfoVecs.lengthZero[indexL] = edgeLen;
		wlcInfoVecs.globalNeighbors[indexL] = idR;
		(wlcInfoVecs.currentNodeEdgeCountVector[idL])++; //right connects to left

		//add edge for right node
		unsigned edgeNumR = wlcInfoVecs.currentNodeEdgeCountVector[idR]; //number of edges on (nodeId = row)	is that entry in cECV
		unsigned indexR = idR*generalParams.maxNeighborCount + edgeNumR;
		wlcInfoVecs.lengthZero[indexR] = edgeLen;
		wlcInfoVecs.globalNeighbors[indexR] = idL;
		(wlcInfoVecs.currentNodeEdgeCountVector[idR])++; //left connects to right

		generalParams.currentEdgeCount++;

	}

	//at this point currentNodeEdgeCountVector holds the number of edges, copy this to
	thrust::copy(wlcInfoVecs.currentNodeEdgeCountVector.begin(), wlcInfoVecs.currentNodeEdgeCountVector.end(), wlcInfoVecs.numOriginalNeighborsNodeVector.begin());
};
