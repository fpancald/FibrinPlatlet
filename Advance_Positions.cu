#include "functor_advance_pos.h"
#include "System.h"
#include "Advance_Positions.h"


double Advance_Positions(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	GeneralParams& generalParams) {


		//At this point, the previous node location is the same as the current node,
		//we can therefore use previous node locations to update nodeLoc.
		 unsigned _seed = rand();
    	thrust::device_vector<double> gaussianData;
    	gaussianData.resize(generalParams.maxNodeCount); //
		thrust::counting_iterator<unsigned> index_sequence_begin(_seed);

    	thrust::transform(thrust::device, index_sequence_begin, index_sequence_begin + (generalParams.maxNodeCount),
        gaussianData.begin(), psrunifgen(-1.0, 1.0));

		thrust::counting_iterator<unsigned> nodeIndexBegin(0);

		thrust::transform(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeIndexBegin,
					nodeInfoVecs.nodeLocX.begin(),
					nodeInfoVecs.nodeLocY.begin(),
					nodeInfoVecs.nodeLocZ.begin())),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeIndexBegin,
					nodeInfoVecs.nodeLocX.begin(),
					nodeInfoVecs.nodeLocY.begin(),
					nodeInfoVecs.nodeLocZ.begin())) + generalParams.maxNodeCount,
			//second vector begin
			thrust::make_zip_iterator(
				thrust::make_tuple(
					gaussianData.begin(),
					nodeInfoVecs.nodeForceX.begin(),
					nodeInfoVecs.nodeForceY.begin(),
					nodeInfoVecs.nodeForceZ.begin())),
			//save result in third vector to test values
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.nodeLocX.begin(),
					nodeInfoVecs.nodeLocY.begin(),
					nodeInfoVecs.nodeLocZ.begin(),
					nodeInfoVecs.nodeVelocity.begin())),
			functor_advance_pos(generalParams.dtTemp,
				generalParams.viscousDamp_Fibrin,
				generalParams.temperature,
				generalParams.kB,
				generalParams.nodeMass,
				generalParams.maxNodeCount,
				thrust::raw_pointer_cast(nodeInfoVecs.isNodeFixed.data())));

		//finally, clear the random data.
        gaussianData.clear();
        gaussianData.shrink_to_fit();

//platelets
unsigned _seedplt = rand();
 thrust::device_vector<double> gaussianPltData;
 gaussianPltData.resize(generalParams.maxPltCount); //
thrust::counting_iterator<unsigned> index_sequence_plt_begin(_seedplt);

 thrust::transform(thrust::device, index_sequence_plt_begin, index_sequence_plt_begin + (generalParams.maxPltCount),
	 gaussianPltData.begin(), psrunifgen(-1.0, 1.0));

thrust::counting_iterator<unsigned> pltIndexBegin(0);

thrust::transform(
 thrust::make_zip_iterator(
	 thrust::make_tuple(
		 pltIndexBegin,
		 pltInfoVecs.pltLocX.begin(),
		 pltInfoVecs.pltLocY.begin(),
		 pltInfoVecs.pltLocZ.begin())),
 thrust::make_zip_iterator(
	 thrust::make_tuple(
		 pltIndexBegin,
		 pltInfoVecs.pltLocX.begin(),
		 pltInfoVecs.pltLocY.begin(),
		 pltInfoVecs.pltLocZ.begin())) + generalParams.maxPltCount,
 //second vector begin
 thrust::make_zip_iterator(
	 thrust::make_tuple(
		 gaussianPltData.begin(),
		 pltInfoVecs.pltForceX.begin(),
		 pltInfoVecs.pltForceY.begin(),
		 pltInfoVecs.pltForceZ.begin())),
 //save result in third vector to test values
 thrust::make_zip_iterator(
	 thrust::make_tuple(
		 pltInfoVecs.pltLocX.begin(),
		 pltInfoVecs.pltLocY.begin(),
		 pltInfoVecs.pltLocZ.begin(),
		 pltInfoVecs.pltVelocity.begin())),
 functor_advance_pos(generalParams.dtTemp,
	 generalParams.viscousDamp_Plt,
	 generalParams.temperature,
	 generalParams.kB,
	 generalParams.pltMass,
	 generalParams.maxPltCount,
	 thrust::raw_pointer_cast(pltInfoVecs.isPltFixed.data())));

//finally, clear the random data.
	 gaussianPltData.clear();
	 gaussianPltData.shrink_to_fit();

	return generalParams.dtTemp;
		//now that nodeLoc is different, we can calculate change and then set previous location
		//to the current location.

}
